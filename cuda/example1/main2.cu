
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__global__
void add(int n, float *x, float *y)
{
    for(int i = 0; i < n; ++i){
        y[i] = x[i] + y[i];
    }
}

int main(int, char **)
{
    const int N = 1 << 20;
    float *x;
    float *y;
    std::cout << "Before alloc" << std::endl;
    //x = new float[N];
    //y = new float[N];
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    std::cout << "Before fill" << std::endl;
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
    return 0;

    std::cout << std::flush;
    std::cout << "Before add" << std::endl;
    std::cout << std::flush;
    add<<<1, 1>>>(N, x, y);

    std::cout << "Before sync" << std::endl;
    hipDeviceSynchronize();
    std::cout << "Before err acc" << std::endl;
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    std::cout << "Before err free" << std::endl;
    hipFree(x);
    hipFree(y);

    return 0;
}

